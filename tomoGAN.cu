#include "tomoGAN.h"

tomoGAN::tomoGAN(uint32 img_n, uint32 img_c, uint32 img_h, uint32 img_w, float *weights_h){
    const uint32 box1_o_sz_h = img_h;
    const uint32 box1_o_sz_w = img_w;
    const uint32 box2_o_sz_h = img_h/2;
    const uint32 box2_o_sz_w = img_w/2;
    const uint32 box3_o_sz_h = img_h/4;
    const uint32 box3_o_sz_w = img_w/4;
    const uint32 intr_o_sz_h = img_h/8;
    const uint32 intr_o_sz_w = img_w/8;

    cudaErrchk( hipMalloc((void **)&input_buf,  img_n * img_c * img_h * img_w * sizeof(uint8)) );
    cudaErrchk( hipMalloc((void **)&output_buf, img_n * 1     * img_h * img_w * sizeof(uint8)) );
    cudaErrchk( hipMalloc((void **)&layer_buf1, img_n * 32    * img_h * img_w * sizeof(float)) );
    cudaErrchk( hipMalloc((void **)&layer_buf2, img_n * 64    * img_h * img_w * sizeof(float)) );
    cudaErrchk( hipMalloc((void **)&box1_out_buf, img_n * 32  * box1_o_sz_h * box1_o_sz_w * sizeof(float)) );
    cudaErrchk( hipMalloc((void **)&box2_out_buf, img_n * 64  * box2_o_sz_h * box2_o_sz_w * sizeof(float)) );
    cudaErrchk( hipMalloc((void **)&box3_out_buf, img_n * 128 * box3_o_sz_h * box3_o_sz_w * sizeof(float)) );

    uint32 w_acc_sz = 0;
    for(auto i = 0; i <= 18; i++){
        auto w_sz = conv_sz[i] * conv_sz[i] * conv_ch[i] * n_conv[i];
        cudaErrchk( hipMalloc((void **)&(weights[i]), w_sz * sizeof(float)) );
        cudaErrchk( hipMemcpy(weights[i], weights + w_acc_sz, w_sz * sizeof(float), hipMemcpyHostToDevice) );
        w_acc_sz += w_sz;
    }

}

tomoGAN::~tomoGAN(){
    hipFree(input_buf);
    hipFree(output_buf);
    hipFree(layer_buf1);
    hipFree(layer_buf2);
    hipFree(box1_out_buf);
    hipFree(box2_out_buf);
    hipFree(box3_out_buf);
}

void tomoGAN::inference(uint8 *img_in, uint8 *img_out){
    
}