#include "hip/hip_runtime.h"
#include "fbp_cuda.h"
#include "kernels.cuh"
#include <stdio.h>
#include <iostream>

using namespace std;

fbp::fbp(float* theta_, float center_, int Ntheta_, int Nz_, int N_){
    N = N_;
    Ntheta = Ntheta_;
    Nz = Nz_;
    center = center_;

    // USFFT parameters
    float eps = 1e-3;
    mu = -log(eps) / (2 * N * N);
    M = ceil(2 * N * 1 / PI * sqrt(-mu * log(eps) + (mu * N) * (mu * N) / 4));

    // arrays allocation on GPU
    cudaErrchk( hipMalloc((void **)&f, N * N * Nz * sizeof(float2)) );
    cudaErrchk( hipMalloc((void **)&g, N * Ntheta * Nz * sizeof(float2)) );
    cudaErrchk( hipMalloc((void **)&fde, 2 * N * 2 * N * Nz * sizeof(float2)) );
    cudaErrchk( hipMalloc((void **)&fdee, (2 * N + 2 * M) * (2 * N + 2 * M) * Nz * sizeof(float2)) );
    cudaErrchk( hipMalloc((void **)&x, N * Ntheta * sizeof(float)) );
    cudaErrchk( hipMalloc((void **)&y, N * Ntheta * sizeof(float)) );
    cudaErrchk( hipMalloc((void **)&theta, Ntheta * sizeof(float)) );
    cudaErrchk( hipMalloc((void **)&shiftadj, N * sizeof(float2)) );

    // init 2d FFTs
    int ffts[2] = {2 * N, 2 * N};
    int idist = 2 * N * 2 * N;
    int odist = (2 * N + 2 * M) * (2 * N + 2 * M);
    int inembed[2] = {2 * N, 2 * N};
    int onembed[2] = {2 * N + 2 * M, 2 * N + 2 * M};

    auto cufft_ret = hipfftPlanMany(&plan2dadj, 2, ffts, onembed, 1, odist, inembed, 1, idist, HIPFFT_C2C, Nz);
    if(cufft_ret != HIPFFT_SUCCESS){
        printf("CUFFT error: hipfftPlanMany failed, file: %s, line: %d\n", __FILE__, __LINE__);
        return;
    }

    // init 1d FFTs
    ffts[0] = N;
    idist = N;
    odist = N;
    inembed[0] = N;
    onembed[0] = N;
    cufft_ret = hipfftPlanMany(&plan1d, 1, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, Ntheta * Nz);
    if(cufft_ret != HIPFFT_SUCCESS){
        printf("CUFFT error: hipfftPlanMany failed, file: %s, line: %d\n", __FILE__, __LINE__);
        return;
    }
    //init thread blocks and block grids
    BS3d.x = 16;
    BS3d.y = 16;
    GS2d0.x = ceil(N / (float)BS3d.x);
    GS2d0.y = ceil(Ntheta / (float)BS3d.y);

    GS3d0.x = ceil(N / (float)BS3d.x);
    GS3d0.y = ceil(N / (float)BS3d.y);
    GS3d0.z = ceil(Nz / (float)BS3d.z);

    GS3d1.x = ceil(2 * N / (float)BS3d.x);
    GS3d1.y = ceil(2 * N / (float)BS3d.y);
    GS3d1.z = ceil(Nz / (float)BS3d.z);

    GS3d2.x = ceil((2 * N + 2 * M) / (float)BS3d.x);
    GS3d2.y = ceil((2 * N + 2 * M) / (float)BS3d.y);
    GS3d2.z = ceil(Nz / (float)BS3d.z);
    
    GS3d3.x = ceil(N / (float)BS3d.x);
    GS3d3.y = ceil(Ntheta / (float)BS3d.y);
    GS3d3.z = ceil(Nz / (float)BS3d.z);

    // copy angles to gpu
    cudaErrchk( hipMemcpy(theta, theta_, Ntheta * sizeof(float), hipMemcpyHostToDevice));
    
    // compute polar coordinates
    takexy<<<GS2d0, BS3d>>>(x, y, theta, N, Ntheta);
    cudaErrchk(hipPeekAtLastError());

    takeshift<<<ceil(N/1024.0), 1024>>>(shiftadj, (center - N / 2.0), N);
    cudaErrchk(hipPeekAtLastError());
}

fbp::~fbp(){
    hipFree(f);
    hipFree(g);
    hipFree(fde);
    hipFree(fdee);
    hipFree(x);
    hipFree(y);
    hipFree(shiftadj);
    hipfftDestroy(plan2dadj);
    hipfftDestroy(plan1d);
}

void fbp::adj(float2* f_, float2* g_){
    size_t mem_free, mem_total;
    hipMemGetInfo(&mem_free, &mem_total);
    printf("%ld out of %ld bytes are free\n", mem_free, mem_total);

    // copy data, init arrays with 0
    cudaErrchk( hipMemcpy(g, g_, N * Ntheta * Nz * sizeof(float2), hipMemcpyHostToDevice) );
    cudaErrchk( hipMemset(fde,  0, 2 * N * 2 * N * Nz * sizeof(float2)) );
    cudaErrchk( hipMemset(fdee, 0, (2 * N + 2 * M) * (2 * N + 2 * M) * Nz * sizeof(float2)) );
    cudaErrchk( hipMemset(f,    0, N * N * Nz * sizeof(float2)) );

    // 1d FFT
    fftshift1c<<<GS3d3, BS3d>>>(g, N, Ntheta, Nz);
    cudaErrchk(hipPeekAtLastError());

    auto ret = hipfftExecC2C(plan1d, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_FORWARD);
    if(ret != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecC2C Forward failed\n");
        return;
    }

    fftshift1c<<<GS3d3, BS3d>>>(g, N, Ntheta, Nz);
    cudaErrchk(hipPeekAtLastError());

    // shift with respect to given center
    shift<<<GS3d3, BS3d>>>(g, shiftadj, N, Ntheta, Nz);
    cudaErrchk(hipPeekAtLastError());

    // filtering 
    applyfilter<<<GS3d3, BS3d>>>(g, N, Ntheta, Nz);
    cudaErrchk(hipPeekAtLastError());

    // scattering from the polar grid
    scatter<<<GS3d3, BS3d>>>(fdee, g, x, y, M, mu, N, Ntheta, Nz);
    cudaErrchk(hipPeekAtLastError());
    
    // wrap frequencies
    wrapadj<<<GS3d2, BS3d>>>(fdee, N, Nz, M);
    cudaErrchk(hipPeekAtLastError());

    // 2d IFFT
    fftshiftc<<<GS3d2, BS3d>>>(fdee, 2 * N + 2 * M, Nz);
    cudaErrchk(hipPeekAtLastError());

    ret = hipfftExecC2C(plan2dadj, (hipfftComplex *)&fdee[M + M * (2 * N + 2 * M)], (hipfftComplex *)fde, HIPFFT_BACKWARD);
    if(ret != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecC2C inverse failed\n");
        return;
    }
    fftshiftc<<<GS3d1, BS3d>>>(fde, 2 * N, Nz);
    cudaErrchk(hipPeekAtLastError());

    // divide by the USFFT kernel function with unpadding
    unpaddivphi<<<GS3d0, BS3d>>>(f, fde, mu, N, Nz);
    cudaErrchk(hipPeekAtLastError());

    // copy result to cpu
    cudaErrchk( hipMemcpy(f_, f, N * N * Nz * sizeof(float2), hipMemcpyDeviceToHost) );
}
